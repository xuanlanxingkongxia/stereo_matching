#include "hip/hip_runtime.h"
#pragma once
#include "Census_cost.cuh"


Census_cost::Census_cost(Mat* left_,Mat* right_)
	:Matching_cost(left_,right_)
{
/*	code_left = new bool**[height];
	code_right = new bool**[height];
	for (int i = 0; i < height; i++)
	{
		code_left[i] = new bool*[width];
		code_right[i] = new bool*[width];
		for (int j = 0; j < width; j++)
		{
			code_left[i][j] = new bool[patch_size*patch_size];
			code_right[i][j] = new bool[patch_size*patch_size];
		}
	}*/
}

Census_cost::Census_cost(Mat* left_, Mat* right_,Matx33d R_,Vec3d T_)
	:Matching_cost(left_, right_, R_, T_)
{
	code_left = new bool**[height];
	code_right = new bool**[height];
	for (int i = 0; i < height; i++)
	{
		code_left[i] = new bool*[width];
		code_right[i] = new bool*[width];
		for (int j = 0; j < width; j++)
		{
			code_left[i][j] = new bool[patch_size*patch_size];
			code_right[i][j] = new bool[patch_size*patch_size];
		}
	}
}

Census_cost::~Census_cost()
{
/*	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			delete[] code_left[i][j];
			delete[] code_right[i][j];
		}
		delete[] code_left[i];
		delete[] code_right[i];
	}
	delete[] code_left;
	delete[] code_right;*/
}

bool Census_cost::evaluate()
{
/*	int test=ad();
	int margin = (patch_size - 1) / 2;
	for (int i = 0 + margin; i < height - margin; i++)
	{
		for (int j = 0 + margin; j < width - margin; j++)
		{
			encode(i, j, left,right);
		}
	}
	Vec3i medium;
	for (int i = 0 + margin; i < height - margin; i++)
	{
		for (int j = 0 + margin; j < width - margin; j++)
		{
			medium = map(i, j, 255);
			if (medium[0] < width && medium[1] < height && medium[1]>0 && medium[0]>0)
			{
				for (int k = 0; k < depth; k++)
				{
					medium = map(i, j, k);
					if (k < depth && medium[0] < width && medium[1] < height)
					{
						cost[i][j][k] = 0;
						for (int q = 0; q < patch_size*patch_size - 1; q++)
						{
							if (code_left[i][j][q] != code_right[medium[1]][medium[0]][q])
							{
								cost[i][j][k]++;
							}
						}
					}
				}
			}
			else
			{
				cost[i][j][0] = LONG_MIN;
			}
		}
	}
	return true;*/

	setDevice(0);
	if (getCudaEnabledDeviceCount() == 0) {
		cerr << "��OpenCV�����ʱ��û������CUDAģ��" << endl;
		return -1;
	}
	GpuMat left_g(left->rows,left->cols,CV_8UC3);
	left_g.upload(*left);
	GpuMat right_g(right->rows, right->cols, CV_8UC3);
	right_g.upload(*right);
	clock_t start = clock();
	cost_compute(left_g, right_g, cost, height, width, depth, (patch_size - 1) / 2);
	clock_t end = clock();
	cout << "gpu����ʱ��:" <<end - start << endl;
	return true;
}
/*
void Census_cost::encode(int h,int w,Mat* this_left,Mat* this_right)
{
	int margin = (patch_size - 1) / 2;
	for (int i = -margin; i <= margin; i++)
	{
		for (int j = -margin; j <= margin; j++)
		{
			if (*this_left->ptr<uchar>(h + i, w + j) > *this_left->ptr<uchar>(h, w))
			{
				code_left[h][w][(i + margin)*patch_size + j + margin] = true;
			}
			else
			{
				code_left[h][w][(i + margin)*patch_size + j + margin] = false;
			}
			if (*this_right->ptr<uchar>(h + i, w + j) > *this_right->ptr<uchar>(h, w))
			{
				code_right[h][w][(i + margin)*patch_size + j + margin] = true;
			}
			else
			{
				code_right[h][w][(i + margin)*patch_size + j + margin] = false;
			}
		}
	}
	return;
}
*/

__global__ void census(PtrStepSz<uchar3> left, PtrStepSz<uchar3> right, hipPitchedPtr cost, int h, int w, int d, int window,int standard)
{
	char* devptr = (char*)cost.ptr;
	size_t pitch = cost.pitch;
	size_t slicepitch = pitch * w;
	for (int q = 0; q < grid_y ; q++)
	{
		int threadid_x = blockIdx.x * blockDim.x + threadIdx.x;
		int threadid_y = q * blockDim.y + threadIdx.y;
		if (threadid_x + d * standard + window < w && threadid_y < h - window && threadid_x >= window && threadid_y >= window)
		{
			uchar3 colar_l0 = left(threadid_y, threadid_x);
			int* pos = (int*)(devptr + threadid_y * slicepitch + threadid_x * pitch);
			for (int i = 0; i < d; i++)
			{
				uchar3 colar_r0 = right(threadid_y, threadid_x + (int)(i*standard));
				pos[i] = 0;
				float a = 0;
				for (int j = threadid_y - window; j < threadid_y + window; j++)
				{
					for (int k = threadid_x - window; k < threadid_x + window; k++)
					{
						uchar3 colar_l1 = left(j, k);
						uchar3 colar_r1 = right(j, k + (int)(i * standard));
						a += ((colar_l1.x > colar_l0.x) != (colar_r1.x > colar_r0.x));
						a += ((colar_l1.y > colar_l0.y) != (colar_r1.y > colar_r0.y));
						a += ((colar_l1.z > colar_l0.z) != (colar_r1.z > colar_r0.z));
					}
				}
				float ad = abs(colar_l0.x - colar_r0.x) + abs(colar_l0.y - colar_r0.y) + abs(colar_l0.z - colar_r0.z);
//				ad = ad;
				pos[i] = 100 - 100*exp(-a / lamda_census) + 100 - 100 * exp(-ad/lamda_ad);
			}
		}
	}
}

void Census_cost::cost_compute(GpuMat& left, GpuMat& right, int* cost, int h, int w, int d, int window)
{
	hipExtent size=make_hipExtent(d * sizeof(int),w,h);       //the sequence of data stored in cuda is h,w,d(from higher to lower) 
	hipError_t status;
//	size_t size = h * w * d * sizeof(long);
	status = hipMalloc3D(&gpu_cost, size);
	if (status != hipSuccess)
	{
		cout << hipGetErrorString(status) << endl;
	}
	dim3 blocksize(block_x, block_y);
	dim3 gridsize(grid_x, 1);
	clock_t time0 = clock();
	census << <gridsize, blocksize >> > (left, right, gpu_cost, h, w, d, window, T[0]);
	status = hipDeviceSynchronize();
	clock_t time1 = clock();
	if (status != hipSuccess)
	{
		cout << hipGetErrorString(status) << endl;
	}
	/*
	hipMemcpy3DParms pos = { 0 };
	pos.srcPtr = gpu_cost;
	pos.kind = hipMemcpyDeviceToHost;
	pos.extent = make_hipExtent(d * sizeof(int), w, h);
	pos.dstPtr = make_hipPitchedPtr((void*)cost, d * sizeof(int), d, w);
	status = hipMemcpy3D(&pos);
	clock_t time2 = clock();
	if (status != hipSuccess)
	{
		cout << hipGetErrorString(status) << endl;
	}
	*/
	cout << "�����ʱ:" << time1 - time0 << endl;
//	cout << "�����ʱ:" << time2 - time1 << endl;
}