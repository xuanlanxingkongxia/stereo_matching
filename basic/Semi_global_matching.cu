#include "hip/hip_runtime.h"
#include "Semi_global_matching.cuh"



Semi_global_matching::Semi_global_matching(int height_, int width_, int depth_, int* cost_)
	:Post_process(height_,width_,depth_,cost_)
{
	cost_now.create(h, w, CV_16UC1);
}

Semi_global_matching::Semi_global_matching(int height_, int width_, int depth_, GpuMat& left_, GpuMat& right_, hipPitchedPtr gpu_cost)
	: Post_process(height_, width_, depth_, left_, right_, gpu_cost)
{
	cost_now.create(h, w, CV_16UC1);
}

Semi_global_matching::~Semi_global_matching()
{
}

void Semi_global_matching::set_image(GpuMat& left_, GpuMat& right_)
{
	left = left_;
	right = right_;
}

__global__ void compute_min(hipPitchedPtr cost,PtrStepSz<int> depth_min,int h,int w,int d)
{
	char *start = (char*)cost.ptr;
	int pitch = cost.pitch;
	int slicepitch = cost.pitch*w;
	for (int q = 0; q < grid_y; q++)
	{
		int threadid_x = blockIdx.x*blockDim.x + threadIdx.x;
		int threadid_y = q * blockDim.y + threadIdx.y;
		if (threadid_x < w && threadid_y < h)
		{
			int *pos = (int*)(start + threadid_y * slicepitch + threadid_x * pitch);
			int min_ = pos[0];
			int num = 0;
			for (int i = 0; i < d; i++)
			{
				if (pos[i] < min_)
				{
					min_ = pos[i];
					num = i;
				}
			}
			depth_min(threadid_y, threadid_x) = min_;
		}
	}
}

__global__ void semi_global(hipPitchedPtr cost, PtrStepSz<uint> cost_now, PtrStepSz<uchar3> left, PtrStepSz<uchar3> right, int h, int w, int d)
{
	char *start = (char*)cost.ptr;
	int pitch = cost.pitch;
	int slicepitch = cost.pitch * w;
	char *middle;
	int *pos;
	int *pos2;
	for (int q = 0; q < grid_y; q++)
	{
		int threadid_x = blockIdx.x * blockDim.x + threadIdx.x;
		int threadid_y = q * blockDim.y + threadIdx.y;
		if (threadid_x + d < w - 1 && threadid_y < h - 1 && threadid_x>=1 && threadid_y>=1)
		{
			middle = start + threadid_y * slicepitch;
			middle = middle + threadid_x * pitch;
			for (int k = 0; k < d; k++)
			{
				pos = (int*)middle;
				for (int x = -1; x <= 1; x++)
				{
					for (int y = -1; y <= 1; y++)
					{
						if (x == 0 && y == 0)
						{
							continue;
						}
						int d1 = abs(left(threadid_y + y, threadid_x + x).z - left(threadid_y, threadid_x).z) + 
							abs(left(threadid_y + y, threadid_x + x).y - left(threadid_y, threadid_x).y) + 
							abs(left(threadid_y + y, threadid_x + x).x - left(threadid_y, threadid_x).x);
						int d2 = abs(right(threadid_y + y, threadid_x + x + d).z - right(threadid_y, threadid_x + d).z) +
							abs(right(threadid_y + y, threadid_x + x + d).y - right(threadid_y, threadid_x + d).y)
							+ abs(right(threadid_y + y, threadid_x + x + d).x - right(threadid_y, threadid_x + d).x);
						int p1;
						int p2;
						if (d1 <= sgm_d)
						{
							if (d2 <= sgm_d)
							{
								p1 = sgm_p1;
								p2 = sgm_p2;
							}
							else
							{
								p1 = sgm_p1 / sgm_q2;
								p2 = sgm_p2 / sgm_q2;
							}
						}
						else
						{
							if (d2 >= sgm_d)
							{
								p1 = sgm_p1 / sgm_q1;
								p2 = sgm_p2 / sgm_q1;
							}
							else
							{
								p1 = sgm_p1 / sgm_q2;
								p2 = sgm_p2 / sgm_q2;
							}
						}
						pos2 = (int*)(middle + (x + threadid_x) * pitch + (y + threadid_y) * slicepitch);
						int t1 = pos2[k];
//						printf("��%d�У���%d�У���%d�� \n", threadid_y, threadid_x, k);
						int a = (k > 1) ? (k - 1) : 0;
						int t2 = pos2[a] + p1;
						int b = k < (d - 1) ? (k + 1) : (d - 1);
						int t3 = pos2[b] + p1;
						int t4 = cost_now(y + threadid_y, x + threadid_x);
						int min1 = t1 < t2 ? t1 : t2;
						int min2 = t3 < t4 ? t3 : t4;
						int min = min1 < min2 ? min1 : min2;
						float t = -cost_now(threadid_y + y, threadid_x + d + x) + min;
						pos[k] += (int)(t / 8);
					}
				}
			}
		}
	}
}

/*
void Semi_global_matching::compute_min()
{
	for (int i = 0; i < h; i++)
	{
		for (int j = 0; j < w; j++)
		{
			int num = 0;
			long min_ = cost[i * d * w + j * d];
			for (int k = 0; k < min<int>(d, w - k); k++)
			{
				if (cost[i * d * w + j * d + k] < min_)
				{
					min_ = cost[i * d * w + j * d + k];
					num = k;
				}
			}
			depth_now.at<float>(i, j) = min_;
		}
	}
}
*/

void Semi_global_matching::process()
{
	dim3 blocksize(block_x, block_y);
	dim3 grid_size(grid_x, 1);
	for (int turn = 0; turn < 10; turn++)
	{
		compute_min << <blocksize, grid_size >> > (gpu_cost, cost_now, h, w, d);
		hipError_t status;
		status = hipDeviceSynchronize();
		if (status != hipSuccess)
		{
			cout << hipGetErrorString(status)<<" ������С����ʧ��"<<endl;
		}
		semi_global << <blocksize, grid_size >> > (gpu_cost, cost_now, left, right, h, w, d);
		status = hipDeviceSynchronize();
		if (status != hipSuccess)
		{
			cout << hipGetErrorString(status) << " �Ż�ʧ��" << endl;
		}
/*		compute_min();
		for (int i = 1; i < h - 1; i++)
		{
			for (int j = 1; j < w - 1; j++)
			{
				int a = *left->ptr<uchar>(i, j);
				int d1 = abs(*left->ptr<uchar>(i, j) - *right->ptr<uchar>(i, j));
				int d2 = abs(*right->ptr<uchar>(i, j) - *right->ptr<uchar>(i, j));
				int p1;
				int p2;
				if (d1 <= sgm_d)
				{
					if (d2 <= sgm_d)
					{
						p1 = sgm_p1;
						p2 = sgm_p2;
					}
					else
					{
						p1 = sgm_p1 / sgm_q2;
						p2 = sgm_p2 / sgm_q2;
					}
				}
				else
				{
					if (d2 >= sgm_d)
					{
						p1 = sgm_p1 / sgm_q1;
						p2 = sgm_p2 / sgm_q1;
					}
					else
					{
						p1 = sgm_p1 / sgm_q2;
						p2 = sgm_p2 / sgm_q2;
					}
				}
				for (int k = 0; k < d; k++)
				{
					for (int x = -1; x <= 1; x++)
					{
						for (int y = -1; y <= 1; y++)
						{
							if (x == 0 && y == 0)
							{
								continue;
							}
							float t = -depth_now.at<float>(i + x, j + y) + min<int>(min<int>(cost[(i + x) * d * w + (j + y) * d + k],
								cost[(i + x) * d * w + (j + y) * d + (k >= 1 ? k - 1 : k)] + p1)
								, min<int>(cost[(i + x) * d * w + (j + y) * d + (k < d - 1 ? k + 1 : k)] + p1, depth_now.at<float>(i + x, j + y) + p2));
							cost[i * d * w + j * d + k] += t/8;
						}
					}*/
/*					float t = -depth_now.at<float>(i + 1, j) + min(min<float>(cost[i + 1][j][k], cost[i + 1][j][k >= 1 ? k - 1 : k] + p1)
						, min<float>(cost[i + 1][j][k < d - 1 ? k + 1 : k] + p1, depth_now.at<float>(i + 1, j) + p2));
					cost[i][j][k] += 0.25* t;
					t = -depth_now.at<float>(i - 1, j ) + min(min<float>(cost[i + 1][j][k], cost[i - 1][j ][k >= 1 ? k - 1 : k] + p1)
						, min<float>(cost[i - 1][j][k < d - 1 ? k + 1 : k] + p1, depth_now.at<float>(i - 1, j) + p2));
					cost[i][j][k] += 0.25* t;
					t = -depth_now.at<float>(i, j + 1) + min(min<float>(cost[i][j + 1][k], cost[i][j + 1][k >= 1 ? k - 1 : k] + p1)
						, min<float>(cost[i][j + 1][k < d - 1 ? k + 1 : k] + p1, depth_now.at<float>(i, j + 1) + p2));
					cost[i][j][k] += 0.25* t;
					t = -depth_now.at<float>(i - 1, j - 1) + min(min<float>(cost[i - 1][j - 1][k], cost[i - 1][j - 1][k >= 1 ? k - 1 : k] + p1)
						, min<float>(cost[i - 1][j - 1][k < d - 1 ? k + 1 : k] + p1, depth_now.at<float>(i - 1, j - 1) + p2));
					cost[i][j][k] += 0.25* t;
				}
			}
		}*/
	}
}