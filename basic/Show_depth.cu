#include "hip/hip_runtime.h"
#include "Show_depth.cuh"

Show_depth::Show_depth(int height_, int width_, int depth_,int* cost_,Mat& depth__,hipPitchedPtr gpu_cost_)
{
	h = height_;
	w = width_;
	d = depth_;
	depth = depth__;
	gpu_cost = gpu_cost_;
}


Show_depth::~Show_depth()
{
}

__global__ void gpu_add(hipPitchedPtr cost, PtrStepSz<uchar1> img,int w, int h)
{
	int threadid_x = blockIdx.x*blockDim.x + threadIdx.x;
	int threadid_y = blockIdx.y*blockDim.y + threadIdx.y;
	if (threadid_x < w && threadid_y < h && threadid_x >= 0 && threadid_y >= 0)
	{
		int pitch = cost.pitch;
		int slicepitch = pitch * w;
		char* point = (char*)cost.ptr;
		int* pos = (int*)(point + threadid_y * slicepitch + threadid_x * pitch);
		int min = *pos;
		int num = 0;
		for (int i = 0; i < pitch / (sizeof(int)); i++)
		{
			if (pos[i] < min)
			{
				min = pos[i];
				num = i;
			}
		}
		img(threadid_y, threadid_x) = make_uchar1(num);
	}	
}

__global__ void Compute_image(hipPitchedPtr cost, PtrStepSz<uchar1> img)
{
	int threadid_x = blockIdx.x*blockDim.x + threadIdx.x;
	int threadid_y = blockIdx.y*blockDim.y + threadIdx.y;
	int pitch = cost.pitch;
	int slicepitch = pitch * cost.ysize;
	char* point = (char*)cost.ptr;
	int* pos = (int*)(point + threadid_y * slicepitch + threadid_x * pitch);
	int min = *pos;
	int num = 0;
	for (int i = 0; i < pitch / (sizeof(int)); i++)
	{
		if (pos[i] < min)
		{
			min = pos[i];
			num = i;
		}
	}
	img(threadid_y, threadid_x) = make_uchar1(num);
}


/*
void Show_depth::compute_img()
{
	
	img.create(h, w, CV_8UC1);
	for (int i = 0; i < h; i++)
	{
		for (int j = 0; j < w; j++)
		{
			int num = 0;
			long min_ = cost[i * d * w + j * d];
			for (int k = 0; k < min<int>(d ,w - k); k++)
			{
				int flag=0;
				if (i == 100 && j == 500 && k == 100)
				{
					flag++;
				}
				if (cost[i * d * w + j * d + k] < min_)
				{
					min_ = cost[i * d * w + j * d + k];
					num = k;
				}
			}
			img.at<uchar>(i, j) = num;
			//delete the bad point
			if (min_ >= 20)
			{
				img.at<uchar>(i, j) = 0;
			}
			else
			{
				img.at<uchar>(i, j) = num;
			}
		}
	}
	//medianBlur(img,img,5);
}
*/
float Show_depth::error_rate()
{
//	cvtColor(depth, depth, CV_RGB2GRAY);
	float num = 0;
	float err = 0;
	float total = 0;
	cout <<"elemsize is:"<< depth.elemSize()<<endl;
	cout << "elemsize1 is:" << depth.elemSize1() << endl;
	float sum = 0;
	for (int i = 0; i < h; i++)
	{
		for (int j = 0; j < w; j++)
		{
			total++;
			if (depth.at<unsigned short>(i, j) != 0 && img.at<uchar>(i, j) != 0)
			{
				num++;
				float medium1 = depth.at<unsigned short>(i, j) / 256.0;
				float medium2 = img.at<uchar>(i, j);
				sum += medium2 / medium1;
				if (abs(medium1-medium2) >= 3)
				{
					err++;
				}
			}
		}
	}
	cout << "����ϵ��Ϊ��" << sum / num << endl;
	cout << "������Ϊ" << err / num << endl;
	cout << "��������ռȫͼ����Ϊ" << num / total << endl;
	return err / num;
}

void Show_depth::show()
{
	GpuMat gmat(h, w, CV_8UC1);
	img.create(h, w, CV_8UC1);
	dim3 block_size(block_x, block_y);
	dim3 grid_size(grid_x, grid_y);
	gpu_add << <grid_size, block_size >> > (gpu_cost, gmat, w, h);
	hipError_t status;
//	Compute_image << <block_size, grid_size >> > (gpu_cost, gmat);
	status = hipDeviceSynchronize();
	if (status != hipSuccess)
	{
		cout << hipGetErrorString(status);
	}
	gmat.download(img);
	imshow("depth_image",img);
}
