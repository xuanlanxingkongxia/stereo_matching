#include "hip/hip_runtime.h"
#include "Cost_aggregation.cuh"

Cost_aggregation::Cost_aggregation(int height_, int width_, int depth_, GpuMat& left_,GpuMat& right_,hipPitchedPtr gpu_cost)
	:Post_process(height_, width_, depth_, left_, right_,gpu_cost)
{
}


Cost_aggregation::~Cost_aggregation()
{
}

__global__ void initialize_num(int h, int w, int d, int*num)
{
	for (int q = 0; q < grid_y; q++)
	{
		int threadid_x = blockIdx.x * blockDim.x + threadIdx.x;
		int threadid_y = q * blockDim.y + threadIdx.y;
		if (threadid_x < w && threadid_y < h  && threadid_x >= 0 && threadid_y >= 0)
		{
			for (int i = 0; i < d; i++)
			{
				num[threadid_y * w * d + threadid_x * d + i] = 1;
			}
		}
	}
}

__global__ void normlize(hipPitchedPtr cost, int h, int w, int d, int* num)
{
	char* devptr = (char*)cost.ptr;
	size_t pitch = cost.pitch;
	size_t slicepitch = pitch * w;
	for (int q = 0; q < grid_y; q++)
	{
		int threadid_x = blockIdx.x * blockDim.x + threadIdx.x;
		int threadid_y = q * blockDim.y + threadIdx.y;
		int* pos = (int*)(devptr + threadid_y * slicepitch + threadid_x * pitch);
		if (threadid_x < w && threadid_y < h  && threadid_x >= 0 && threadid_y >= 0)
		{
			for (int i = 0; i < d; i++)
			{
				pos[i] /= (float)num[threadid_y * w * d + threadid_x * d + i];
			}
		}
	}
}


__global__ void aggregation(hipPitchedPtr cost, hipPitchedPtr cost2, int h, int w, int d, int* limits,int* num, int* num2,bool direction_h)
{
	int* devptr = (int*)cost.ptr;
	int* devptr2 = (int*)cost2.ptr;
	size_t pitch = cost.pitch / sizeof(int);
	size_t slicepitch = pitch * w;
	for (int q = 0; q < grid_y; q++)
	{
		int threadid_x = blockIdx.x * blockDim.x + threadIdx.x;
		int threadid_y = q * blockDim.y + threadIdx.y;
		if (threadid_x < w && threadid_y < h  && threadid_x >= 0 && threadid_y >= 0)
		{
			for (int i = 0; i < d; i++)
			{
				int sum_num = 0;
				int sum = 0;
				if (direction_h)
				{
					int plus = limits[up * h * w + threadid_y * w + threadid_x];
					int minus = -limits[down * h * w + threadid_y * w + threadid_x];
					for (int k = minus; k <= plus; k++)
					{
						sum = sum + devptr[(threadid_y + k) * w * pitch + threadid_x * pitch + i];
						sum_num = sum_num + num[(threadid_y + k) * w * pitch + threadid_x * pitch + i];
/*						if (threadid_x == 500 && threadid_y == 150 && i == 10)
						{
							printf("��%d�֣�", k);
							printf("sum:%d \n", devptr[threadid_y * w * pitch + threadid_x * pitch + i]);
							printf("sum_origin:%d \n", sum);
							printf("sum_num:%d \n", num[threadid_y * w * pitch + threadid_x * pitch + i]);
							printf("sum_num_origin:%d \n \n", sum_num);
						}*/
					}
				}
				else
				{
					int plus = limits[r * h * w + threadid_y * w + threadid_x];
					int minus = -limits[l * h * w + threadid_y * w + threadid_x];
					for (int k = minus; k <= plus; k++)
					{
						sum = sum + devptr[threadid_y * w * pitch +(threadid_x + k ) * pitch + i];
						sum_num = sum_num + num[threadid_y * w * pitch + (threadid_x + k) * pitch + i];
					}
				}
				devptr2[threadid_y * w * pitch + threadid_x * pitch + i] = sum;
				num2[threadid_y * w * pitch + threadid_x * pitch + i] = sum_num;
/*				if (threadid_x == 500 && threadid_y == 150 && i==10)
				{
					printf("sum:%d \n", devptr[threadid_y * w * pitch + threadid_x * pitch + i]);
					printf("sum_origin:%d \n", sum);
					printf("sum_num:%d \n", num[threadid_y * w * pitch + threadid_x * pitch + i]);
					printf("sum_num_origin:%d \n \n", sum_num);
				}*/
			}
		}
	}
}

__global__ void compute_limits(PtrStepSz<uchar3> left, int h, int w, int dir_x, int dir_y, int* distance)
{
	for (int q = 0; q < grid_y; q++)
	{
		int threadid_x = blockIdx.x * blockDim.x + threadIdx.x;
		int threadid_y = q * blockDim.y + threadIdx.y;
		if (threadid_x < w && threadid_y < h  && threadid_x >= 0 && threadid_y >= 0)
		{
			uchar3 colar0 = left(threadid_y, threadid_x);
			int i = 1;
			for (; i < L1; i++)
			{
				if (i * dir_x + threadid_x >= w || i * dir_y + threadid_y >= h || i * dir_x + threadid_x < 0 || i * dir_y + threadid_y < 0)
				{
					break;
				}
				uchar3 colar1 = left(threadid_y + i * dir_y, threadid_x + i * dir_x);
				int diff_x = abs(colar1.x - colar0.x);
				int diff_y = abs(colar1.y - colar0.y);
				int diff_z = abs(colar1.z - colar0.z);
				if (diff_x > E1 || diff_y > E1 || diff_z > E1)
				{
					break;
				}
			}
			for (; i >= L1 && i < L2; i++)
			{
				if (i * dir_x + threadid_x >= w || i * dir_y + threadid_y >= h || i * dir_x + threadid_x < 0 || i * dir_y + threadid_y < 0)
				{
					break;
				}
				uchar3 colar1 = left(threadid_y + i * dir_y, threadid_x + i * dir_x);
				int diff_x = abs(colar1.x - colar0.x);
				int diff_y = abs(colar1.y - colar0.y);
				int diff_z = abs(colar1.z - colar0.z);
				if (diff_x > E2 || diff_y > E2 || diff_z > E2)
				{
					break;
				}
			}
			distance[threadid_y*w + threadid_x] = i - 1;
		}
	}
}



void Cost_aggregation::process()
{
	dim3 blocksize(block_x,block_y);
	dim3 gridsize(grid_x,1);
	hipError_t status;
	size_t size = h * w * sizeof(int);
	hipMalloc((void**)&(limits), 4 * size);
	compute_limits << <gridsize, blocksize >> > (left, h, w, 1, 0, &limits[r*h*w]);
	hipDeviceSynchronize();
	compute_limits << <gridsize, blocksize >> > (left, h, w, -1, 0, &limits[l*h*w]);
	hipDeviceSynchronize();
	compute_limits << <gridsize, blocksize >> > (left, h, w, 0, 1, &limits[up*h*w]);
	hipDeviceSynchronize();
	compute_limits << <gridsize, blocksize >> > (left, h, w, 0, -1, &limits[down*h*w]);
	status = hipDeviceSynchronize();
	hipPitchedPtr gpu_cost2;
	hipExtent cost_size = make_hipExtent(d * sizeof(int), w, h);
	hipMalloc3D(&gpu_cost2, cost_size);
	int* num;
	int* num2;
	size_t size2 = h * w * d * sizeof(int);
	hipMalloc((void**)&num, size2);
	hipMalloc((void**)&num2, size2);
	initialize_num<<<gridsize,blocksize>>>(h, w, d, num);
	hipDeviceSynchronize();
	if (status != hipSuccess)
	{
		cout << hipGetErrorString(status) << "�������Ʋ���";
	}
	aggregation << <gridsize, blocksize >> > (gpu_cost, gpu_cost2, h, w, d, limits, num, num2, true);
	status = hipDeviceSynchronize();
	if (status != hipSuccess)
	{
		cout << hipGetErrorString(status) << "����ۺϲ���";
	}
	aggregation << <gridsize, blocksize >> > (gpu_cost2, gpu_cost, h, w, d, limits, num2, num, false);
	status = hipDeviceSynchronize();
	if (status != hipSuccess)
	{
		cout << hipGetErrorString(status) << "����ۺϲ���";
	}
	normlize << <gridsize, blocksize >> > (gpu_cost, h, w, d, num);
	status = hipDeviceSynchronize();
	if (status != hipSuccess)
	{
		cout << hipGetErrorString(status) << "����ۺϲ���";
	}
	initialize_num << <gridsize, blocksize >> > (h, w, d, num);
	hipDeviceSynchronize();
	aggregation << <gridsize, blocksize >> > (gpu_cost, gpu_cost2, h, w, d, limits, num, num2, false);
	status = hipDeviceSynchronize();
	aggregation << <gridsize, blocksize >> > (gpu_cost2, gpu_cost, h, w, d, limits, num2, num, true);
	status = hipDeviceSynchronize();
	if (status != hipSuccess)
	{
		cout << hipGetErrorString(status) << "����ۺϲ���";
	}
	normlize << <gridsize, blocksize >> > (gpu_cost, h, w, d, num);
	hipDeviceSynchronize();

	hipFree(limits);
	hipFree(num);
	hipFree(num2);
}

