#include "hip/hip_runtime.h"
#include "adcensus.cuh"
#include<vector>
#include <time.h>
#define block_x 16
#define block_y 16
#define grid_x 78
#define grid_y 24


/*
__global__ void census(PtrStepSz<uchar3> left, PtrStepSz<uchar3> right, hipPitchedPtr cost, int h, int w, int d, int window)
{
	char* devptr = (char*)cost.ptr;
	size_t pitch = cost.pitch;
	size_t slicepitch = pitch * w;
	for (int q = 0; q < grid_y ; q++)
	{
		int threadid_x = blockIdx.x * blockDim.x + threadIdx.x;
		int threadid_y = q * blockDim.y + threadIdx.y;
		if (threadid_x + d + window < w && threadid_y < h - window && threadid_x >= window && threadid_y >= window)
		{
			uchar3 colar_l0 = left(threadid_y, threadid_x);
			int* pos = (int*)(devptr + threadid_y * slicepitch + threadid_x * pitch);
			for (int i = 0; i < d; i++)
			{
				uchar3 colar_r0 = right(threadid_y, threadid_x + i);
				pos[i] = 0;
				for (int j = threadid_y - window; j < threadid_y + window; j++)
				{
					for (int k = threadid_x - window; k < threadid_x + window; k++)
					{
						uchar3 colar_l1 = left(j, k);
						uchar3 colar_r1 = right(j, k + i);
						pos[i] += (int)((colar_l1.x > colar_l0.x) != (colar_r1.x > colar_r0.x));
						pos[i] += (int)((colar_l1.y > colar_l0.y) != (colar_r1.y > colar_r0.y));
						pos[i] += (int)((colar_l1.z > colar_l0.z) != (colar_r1.z > colar_r0.z));
					}
				}
			}
		}
	}
}

namespace cuda_cen
{
	void cost_compute(GpuMat& left, GpuMat& right, int* cost, int h, int w, int d, int window)
	{
		hipPitchedPtr gpu_cost;
		hipExtent size=make_hipExtent(d * sizeof(int),w,h);       //the sequence of data stored in cuda is h,w,d(from higher to lower) 
		hipError_t status;
//		size_t size = h * w * d * sizeof(long);
		status = hipMalloc3D(&gpu_cost, size);
		if (status != hipSuccess)
		{
			cout << hipGetErrorString(status) << endl;
		}
		dim3 blocksize(block_x, block_y);
		dim3 gridsize(grid_x, 1);
		clock_t time0 = clock();
		census << <gridsize, blocksize >> > (left, right, gpu_cost, h, w, d, window);
		status = hipDeviceSynchronize();
		clock_t time1 = clock();
		if (status != hipSuccess)
		{
			cout << hipGetErrorString(status) << endl;
		}
		hipMemcpy3DParms pos = { 0 };
		pos.srcPtr = gpu_cost;
		pos.kind = hipMemcpyDeviceToHost;
		pos.extent = make_hipExtent(d * sizeof(int), w, h);
		pos.dstPtr = make_hipPitchedPtr((void*)cost, d * sizeof(int), d, w);
		status = hipMemcpy3D(&pos);
		clock_t time2 = clock();
		if (status != hipSuccess)
		{
			cout << hipGetErrorString(status) << endl;
		}
		cout << "�����ʱ:" << time1 - time0 << endl;
		cout << "�����ʱ:" << time2 - time1 << endl;
	}
}
*/
